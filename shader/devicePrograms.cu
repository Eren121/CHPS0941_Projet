#include "hip/hip_runtime.h"

// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#define DEVICE 1
#include <optix_device.h>

#include "../src/LaunchParams.h"

  
  /*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
  extern "C" __constant__ LaunchParams optixLaunchParams;

  static __forceinline__ __device__
  void *unpackPointer( uint32_t i0, uint32_t i1 )
  {
    const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr ); 
    return ptr;
  }

  static __forceinline__ __device__
  void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
  {
    const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD()
  { 
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
  }
  
  //------------------------------------------------------------------------------
  // closest hit and anyhit programs for radiance-type rays.
  //
  // Note eventually we will have to create one pair of those for each
  // ray type and each geometry type we want to render; but this
  // simple example doesn't use any actual geometries yet, so we only
  // create a single, dummy, set of them (we do have to have at least
  // one group of them to set up the SBT)
  //------------------------------------------------------------------------------
  
  //------------------ MESH ----------------------------

  extern "C" __global__ void __closesthit__mesh_radiance()
  {
    const sbtData &sbt
      = *(const sbtData*)optixGetSbtDataPointer();

    vec3f &prd = *(vec3f*)getPRD<vec3f>();
    prd = vec3f(sbt.meshData.kd.x, sbt.meshData.kd.y, sbt.meshData.kd.z);
  }
  
  extern "C" __global__ void __anyhit__mesh_radiance()
  { /*! for this simple example, this will remain empty */ }


// ------------- Volume ----------------------------
#define stepSize_current optixLaunchParams.frame.sampler
  __device__ void swap(float &a, float &b) {
    float tmp = a;
    a = b;
    b = tmp;
  }
  __device__ bool inVolume(const VolumetricCube &data, const vec3f &pi){
        bool res = false;
        const vec3f min = data.center - data.size/2.0f;
        const vec3f max = data.center + data.size/2.0f;
        if(( pi.x <= max.x && pi.x >= min.x)&&
                ( pi.y <= max.y && pi.y >= min.y) &&
                ( pi.z <= max.z && pi.z >= min.z))
            res = true;
        return res;
  }
  
  __device__ void mip(){
      const VolumetricCube& sbtData
       = *(const VolumetricCube*)optixGetSbtDataPointer();
     const int   primID = optixGetPrimitiveIndex();
     intersection_time time;
     time.tmin.uitmin = optixGetAttribute_0();
     time.tmax.uitmax = optixGetAttribute_1();
     //Gather information
     vec3f ro = optixGetWorldRayOrigin();
     vec3f rd = optixGetWorldRayDirection();
     vec3f& prd = *(vec3f*)getPRD<vec3f>();
     vec3f sizeP = vec3f(sbtData.sizePixel.x, sbtData.sizePixel.y, sbtData.sizePixel.z);

     //Ray
     vec3f point_in = ro + time.tmin.ftmin * rd ;
     vec3f point_out = ro + time.tmax.ftmax * rd;
     vec3f ray_world = point_out - point_in;

     vec3f step_vector_tex = normalize(ray_world) * stepSize_current;
     float current_ray_length = norme(ray_world);

     vec3f current_pos_tex = point_in;
     float current_max = 0.0f;
     float current_intensity = 0.0f;

     //MIP
     prd = vec3f(0.0f);
     while(current_ray_length > 0.0f){
        vec3f pos_tex = (current_pos_tex - sbtData.center + sbtData.size / 2.0f) / sbtData.size;
        current_intensity = tex3D<float>(sbtData.tex,pos_tex.x,pos_tex.y,pos_tex.z);


        if( current_intensity >= optixLaunchParams.frame.minIntensity && current_intensity <= optixLaunchParams.frame.maxIntensity){
            if( current_intensity > current_max )
                current_max = current_intensity;

            if( current_max == 1.0f){
                prd = vec3f(1.0f);
                break;
            }
            prd = vec3f(current_max);
        }
        current_pos_tex = current_pos_tex + step_vector_tex;
        current_ray_length -= stepSize_current;
     }
  }

  extern "C" __global__ void __closesthit__volume_radiance(){
     /* const VolumetricCube& data
       = (*(const sbtData*)optixGetSbtDataPointer()).volumeData;*/
      const sbtData &sbt
      = *(const sbtData*)optixGetSbtDataPointer();

    vec3f &prd = *(vec3f*)getPRD<vec3f>();
    prd = sbt.volumeData.color;
  }


  extern "C" __global__ void __anyhit__volume_radiance()
  {
  }


  extern "C" __global__ void __intersection__volume() {
      const VolumetricCube& sbt
          = (*(const sbtData*)optixGetSbtDataPointer()).volumeData;
      vec3f ro = optixGetWorldRayOrigin();
      vec3f rayDir = optixGetWorldRayDirection();
      vec3f min, max;
      min = sbt.center - sbt.size / 2;
      max = sbt.center + sbt.size / 2;

      float tmin = (min.x - ro.x) / rayDir.x;
      float tmax = (max.x - ro.x) / rayDir.x;

      if (tmin > tmax) swap(tmin, tmax);

      float tymin = (min.y - ro.y) / rayDir.y;
      float tymax = (max.y - ro.y) / rayDir.y;

      if (tymin > tymax) swap(tymin, tymax);

      //Rayon en dehors du cube (normalement impossible )
      if ((tmin > tymax) || (tymin > tmax))

        return ;

      if (tymin > tmin)
          tmin = tymin;

      if (tymax < tmax)
          tmax = tymax;

      float tzmin = (min.z - ro.z) / rayDir.z;
      float tzmax = (max.z - ro.z) / rayDir.z;

      if (tzmin > tzmax) swap(tzmin, tzmax);

      //Rayon en dehors du cube (normalement impossible )
      if ((tmin > tzmax) || (tzmin > tmax))
        return ;

      if (tzmin > tmin)
          tmin = tzmin;

      if (tzmax < tmax)
          tmax = tzmax;
      if (tmin > tmax) swap(tmin, tmax);
      intersection_time time;

      time.tmin.ftmin = tmin;
      time.tmax.ftmax = tmax;
      optixReportIntersection(tmin, 1,time.tmin.uitmin, time.tmax.uitmax);
  }
  
  //------------------------------------------------------------------------------
  // miss program that gets called for any ray that did not have a
  // valid intersection
  //
  // as with the anyhit/closest hit programs, in this example we only
  // need to have _some_ dummy function to set up a valid SBT
  // ------------------------------------------------------------------------------
  
  extern "C" __global__ void __miss__radiance()
  {
    vec3f &prd = *(vec3f*)getPRD<vec3f>();
    // set to constant white as background color
    prd = vec3f(0.f,0.f,0.f);
  }

  //------------------------------------------------------------------------------
  // ray gen program - the actual rendering happens in here
  //------------------------------------------------------------------------------
  extern "C" __global__ void __raygen__renderFrame()
  {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    if( ix == 0 && iy == 0)
      printf("__raygen__renderFrame\n");
    const auto &camera = optixLaunchParams.camera;

    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    vec3f pixelColorPRD = vec3f(0.f);

    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );

    // normalized screen plane position, in [0,1]^2
    const vec2f screen(vec2f(ix+.5f,iy+.5f)
                       / vec2f(optixLaunchParams.frame.size.x , optixLaunchParams.frame.size.y));
    
    // generate ray direction
    vec3f rayDir = normalize(camera.direction
                             + (screen.x - 0.5f) * camera.horizontal
                             + (screen.y - 0.5f) * camera.vertical);

    const vec3f ro = camera.position;
    const vec3f rd = rayDir;

   
    const float3 cp = make_float3(ro.x ,ro.y,ro.z);
    const float3 rdf3 = make_float3(rd.x,rd.y,rd.z);
    optixTrace(optixLaunchParams.traversable,
               cp,
               rdf3,
               0.f,    // tmin
               1e20f,  // tmax
               0.0f,   // rayTime
               OptixVisibilityMask( 255 ),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,             // SBT offset
               RAY_TYPE_COUNT,               // SBT stride
               SURFACE_RAY_TYPE,             // missSBTIndex 
               u0, u1 );

    const int r = int(255.99f*pixelColorPRD.x);
    const int g = int(255.99f*pixelColorPRD.y);
    const int b = int(255.99f*pixelColorPRD.z);
    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000
      | (r<<0) | (g<<8) | (b<<16);

    // and write to frame buffer ...
    const uint32_t fbIndex = ix+iy*optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
  }
